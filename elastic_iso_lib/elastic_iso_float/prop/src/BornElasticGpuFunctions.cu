#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>
#include "BornElasticGpuFunctions.h"
#include "kernelsGpuElastic.cu"
#include "cudaErrors.cu"
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <ctime>
#include <stdio.h>
#include <assert.h>
#include <stdexcept>


/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
bool getGpuInfo(std::vector<int> gpuList, int info, int deviceNumberInfo){

		int nDevice, driver;
		hipGetDeviceCount(&nDevice);

		if (info == 1){

				std::cout << " " << std::endl;
				std::cout << "-------------------------------------------------------------------" << std::endl;
				std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
				std::cout << "-------------------------------------------------------------------" << std::endl;

				// Number of devices
				std::cout << "Number of requested GPUs: " << gpuList.size() << std::endl;
				std::cout << "Number of available GPUs: " << nDevice << std::endl;
				std::cout << "Id of requested GPUs: ";
				for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
					if (iGpu<gpuList.size()-1){std::cout << gpuList[iGpu] << ", ";}
					else{ std::cout << gpuList[iGpu] << std::endl;}
				}

				// Driver version
				std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl; // Driver

				// Get properties
				hipDeviceProp_t dprop;
				hipGetDeviceProperties(&dprop,deviceNumberInfo);

				// Display
				std::cout << "Name: " << dprop.name << std::endl;
				std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
				std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
				std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
				std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
				std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
				std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
				std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
				std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
				std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
				std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
				if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
				if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
				if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
				if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
				if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
				if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

				std::cout << "-------------------------------------------------------------------" << std::endl;
				std::cout << " " << std::endl;
		}

		// Check that the number of requested GPU is less or equal to the total number of available GPUs
		if (gpuList.size()>nDevice) {
			std::cout << "**** ERROR [getGpuInfo]: Number of requested GPU greater than available GPUs ****" << std::endl;
			return false;
		}

		// Check that the GPU numbers in the list are between 0 and nGpu-1
		for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
			if (gpuList[iGpu]<0 || gpuList[iGpu]>nDevice-1){
				std::cout << "**** ERROR [getGpuInfo]: One of the element of the GPU Id list is not a valid GPU Id number ****" << std::endl;
				return false;
			}
		}

		return true;
}


void initBornGpu(float dz, float dx, int nz, int nx, int nts, float dts, int sub, int minPad, int blockSize, float alphaCos, int nGpu, int iGpuId, int iGpuAlloc){

		// Set GPU
		hipSetDevice(iGpuId);

		// Host variables
		host_nz = nz;
		host_nx = nx;
		host_nts = nts;
		host_sub = sub;
		host_ntw = (nts - 1) * sub + 1;

		/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
		// Only one GPU will perform the following
		if (iGpuId == iGpuAlloc) {

				// Time slices for FD stepping for each wavefield
				dev_p0_vx       = new float*[nGpu];
				dev_p0_vz       = new float*[nGpu];
				dev_p0_sigmaxx  = new float*[nGpu];
				dev_p0_sigmazz  = new float*[nGpu];
				dev_p0_sigmaxz  = new float*[nGpu];

				dev_p1_vx       = new float*[nGpu];
				dev_p1_vz       = new float*[nGpu];
				dev_p1_sigmaxx  = new float*[nGpu];
				dev_p1_sigmazz  = new float*[nGpu];
				dev_p1_sigmaxz  = new float*[nGpu];

				dev_temp1    = new float*[nGpu];

				// Data and model
				dev_sourceRegDtw_vx = new float*[nGpu];
				dev_sourceRegDtw_vz = new float*[nGpu];
				dev_sourceRegDtw_sigmaxx = new float*[nGpu];
				dev_sourceRegDtw_sigmazz = new float*[nGpu];
				dev_sourceRegDtw_sigmaxz = new float*[nGpu];
				dev_dataRegDts_vx = new float*[nGpu];
				dev_dataRegDts_vz = new float*[nGpu];
				dev_dataRegDts_sigmaxx = new float*[nGpu];
				dev_dataRegDts_sigmazz = new float*[nGpu];
				dev_dataRegDts_sigmaxz = new float*[nGpu];

				// Source and receivers
				dev_sourcesPositionRegCenterGrid = new int*[nGpu];
				dev_sourcesPositionRegXGrid = new int*[nGpu];
				dev_sourcesPositionRegZGrid = new int*[nGpu];
				dev_sourcesPositionRegXZGrid = new int*[nGpu];
				dev_receiversPositionRegCenterGrid = new int*[nGpu];
				dev_receiversPositionRegXGrid = new int*[nGpu];
				dev_receiversPositionRegZGrid = new int*[nGpu];
				dev_receiversPositionRegXZGrid = new int*[nGpu];

				// Scaled velocity
				dev_rhoxDtw = new float*[nGpu]; // Precomputed scaling dtw / rho_x
				dev_rhozDtw = new float*[nGpu]; // Precomputed scaling dtw / rho_z
				dev_lamb2MuDtw = new float*[nGpu]; // Precomputed scaling (lambda + 2*mu) * dtw
				dev_lambDtw = new float*[nGpu]; // Precomputed scaling lambda * dtw
				dev_muxzDtw = new float*[nGpu]; // Precomputed scaling mu_xz * dtw

				// Streams for saving the wavefield and time slices
				compStream = new hipStream_t[nGpu];
				transferStream = new hipStream_t[nGpu];
				pin_wavefieldSlice = new float*[nGpu];
				dev_wavefieldDts_left = new float*[nGpu];
				dev_wavefieldDts_right = new float*[nGpu];
				dev_pStream = new float*[nGpu];

        // Pointers specific to Born operator
        dev_ssVxLeft  = new float*[nGpu];
        dev_ssVxRight = new float*[nGpu];
        dev_ssVzLeft  = new float*[nGpu];
        dev_ssVzRight = new float*[nGpu];
        dev_ssSigmaxxLeft  = new float*[nGpu];
        dev_ssSigmaxxRight = new float*[nGpu];
        dev_ssSigmazzLeft  = new float*[nGpu];
        dev_ssSigmazzRight = new float*[nGpu];
        dev_ssSigmaxzLeft  = new float*[nGpu];
        dev_ssSigmaxzRight = new float*[nGpu];

        dev_drhox = new float*[nGpu];
        dev_drhoz = new float*[nGpu];
        dev_dlame = new float*[nGpu];
        dev_dmu   = new float*[nGpu];
        dev_dmuxz = new float*[nGpu];

        dev_wavefieldVx = new float*[nGpu];
        dev_wavefieldVz = new float*[nGpu];

		}

		/**************************** COMPUTE DERIVATIVE COEFFICIENTS ************************/
		float zCoeff[COEFF_SIZE];
		float xCoeff[COEFF_SIZE];

		zCoeff[0] = 1.196289062541883 / dz;
		zCoeff[1] = -0.079752604188901 / dz;
		zCoeff[2] = 0.009570312506634 / dz;
		zCoeff[3] = -6.975446437140719e-04 / dz;

		xCoeff[0] = 1.196289062541883 / dx;
		xCoeff[1] = -0.079752604188901 / dx;
		xCoeff[2] = 0.009570312506634 / dx;
		xCoeff[3] = -6.975446437140719e-04 / dx;


		/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
		// Time interpolation filter length/half length
		int hInterpFilter = host_sub + 1;
		int nInterpFilter = 2 * hInterpFilter;

		// Check the subsampling coefficient is smaller than the maximum allowed
		if (sub>=SUB_MAX){
				std::cout << "**** ERROR: Subsampling parameter too high ****" << std::endl;
				assert (1==2);
		}

		// Allocate and fill interpolation filter
		float interpFilter[nInterpFilter];
		for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
				interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
				interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
				interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(float(host_ntw)/float(host_nts)));
				interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(float(host_ntw)/float(host_nts)));
		}

		/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
		if (minPad>=PAD_MAX){
				std::cout << "**** ERROR: Padding value is too high ****" << std::endl;
				assert (1==2);
		}
		float cosDampingCoeff[minPad];

		// Cosine padding
		for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
				float arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
				arg = alphaCos + (1.0-alphaCos) * cos(arg);
				cosDampingCoeff[iFilter-FAT] = arg;
		}

		// Check that the block size is consistent between parfile and "varDeclare.h"
		if (blockSize != BLOCK_SIZE) {
				std::cout << "**** ERROR: Block size for time stepper is not consistent with parfile ****" << std::endl;
				assert (1==2);
		}

		/**************************** COPY TO CONSTANT MEMORY *******************************/
		// Laplacian coefficients
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_zCoeff), zCoeff, COEFF_SIZE*sizeof(float), 0, hipMemcpyHostToDevice)); // Copy derivative coefficients to device
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCoeff), xCoeff, COEFF_SIZE*sizeof(float), 0, hipMemcpyHostToDevice));

		// Time interpolation filter
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_interpFilter), interpFilter, nInterpFilter*sizeof(float), 0, hipMemcpyHostToDevice)); // Filter

		// Cosine damping parameters
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(float), 0, hipMemcpyHostToDevice)); // Array for damping
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(float), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

		// FD parameters
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
  	float inv_dts = 1.0/(2.0 * dts);
  	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_dts_inv), &inv_dts, sizeof(float), 0, hipMemcpyHostToDevice)); // Inverse of the time-source sampling

}


void allocateBornElasticGpu(float *rhoxDtw, float *rhozDtw, float *lamb2MuDt, float *lambDtw, float *muxzDt, int iGpu, int iGpuId, int useStreams){

	// Get GPU number
	hipSetDevice(iGpuId);

	// Allocate scaled elastic parameters to device
	cuda_call(hipMalloc((void**) &dev_rhoxDtw[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_rhozDtw[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_lamb2MuDtw[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_lambDtw[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_muxzDtw[iGpu], host_nz*host_nx*sizeof(float)));

	// Copy scaled elastic parameters to device
	cuda_call(hipMemcpy(dev_rhoxDtw[iGpu], rhoxDtw, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
	cuda_call(hipMemcpy(dev_rhozDtw[iGpu], rhozDtw, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
	cuda_call(hipMemcpy(dev_lamb2MuDtw[iGpu], lamb2MuDt, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
	cuda_call(hipMemcpy(dev_lambDtw[iGpu], lambDtw, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
	cuda_call(hipMemcpy(dev_muxzDtw[iGpu], muxzDt, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));

	// Allocate wavefield time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p0_vx[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p0_vz[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p0_sigmaxx[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p0_sigmazz[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p0_sigmaxz[iGpu], host_nz*host_nx*sizeof(float)));

	cuda_call(hipMalloc((void**) &dev_p1_vx[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p1_vz[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p1_sigmaxx[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p1_sigmazz[iGpu], host_nz*host_nx*sizeof(float)));
	cuda_call(hipMalloc((void**) &dev_p1_sigmaxz[iGpu], host_nz*host_nx*sizeof(float)));

  //Allocating memory specific to Born operator
  cuda_call(hipMalloc((void**) &dev_ssVxLeft[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssVxRight[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssVzLeft[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssVzRight[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmaxxLeft[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmaxxRight[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmazzLeft[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmazzRight[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmaxzLeft[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_ssSigmaxzRight[iGpu], host_nz*host_nx*sizeof(float)));

  cuda_call(hipMalloc((void**) &dev_drhox[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_drhoz[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_dlame[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_dmu[iGpu], host_nz*host_nx*sizeof(float)));
  cuda_call(hipMalloc((void**) &dev_dmuxz[iGpu], host_nz*host_nx*sizeof(float)));

  //If streams are used, allocate wavefield memory on the device
  if(useStreams == 0){
      cuda_call(hipMalloc((void**) &dev_wavefieldVx[iGpu], host_nz*host_nx*host_nts*sizeof(float)));
      cuda_call(hipMalloc((void**) &dev_wavefieldVz[iGpu], host_nz*host_nx*host_nts*sizeof(float)));
  }
}

void deallocateBornElasticGpu(int iGpu, int iGpuId, int useStreams){
	hipSetDevice(iGpuId); // Set device number on GPU cluster

	// Deallocate scaled elastic params
	cuda_call(hipFree(dev_rhoxDtw[iGpu]));
	cuda_call(hipFree(dev_rhozDtw[iGpu]));
	cuda_call(hipFree(dev_lamb2MuDtw[iGpu]));
	cuda_call(hipFree(dev_lambDtw[iGpu]));
	cuda_call(hipFree(dev_muxzDtw[iGpu]));

	// Deallocate wavefield slices
	cuda_call(hipFree(dev_p0_vx[iGpu]));
	cuda_call(hipFree(dev_p0_vz[iGpu]));
	cuda_call(hipFree(dev_p0_sigmaxx[iGpu]));
	cuda_call(hipFree(dev_p0_sigmazz[iGpu]));
	cuda_call(hipFree(dev_p0_sigmaxz[iGpu]));

	cuda_call(hipFree(dev_p1_vx[iGpu]));
	cuda_call(hipFree(dev_p1_vz[iGpu]));
	cuda_call(hipFree(dev_p1_sigmaxx[iGpu]));
	cuda_call(hipFree(dev_p1_sigmazz[iGpu]));
	cuda_call(hipFree(dev_p1_sigmaxz[iGpu]));

	//Deallocate memory specific to Born operator
	cuda_call(hipFree(dev_ssVxLeft[iGpu]));
	cuda_call(hipFree(dev_ssVxRight[iGpu]));
	cuda_call(hipFree(dev_ssVzLeft[iGpu]));
	cuda_call(hipFree(dev_ssVzRight[iGpu]));
	cuda_call(hipFree(dev_ssSigmaxxLeft[iGpu]));
	cuda_call(hipFree(dev_ssSigmaxxRight[iGpu]));
	cuda_call(hipFree(dev_ssSigmazzLeft[iGpu]));
	cuda_call(hipFree(dev_ssSigmazzRight[iGpu]));
	cuda_call(hipFree(dev_ssSigmaxzLeft[iGpu]));
	cuda_call(hipFree(dev_ssSigmaxzRight[iGpu]));

	cuda_call(hipFree(dev_drhox[iGpu]));
	cuda_call(hipFree(dev_drhoz[iGpu]));
	cuda_call(hipFree(dev_dlame[iGpu]));
	cuda_call(hipFree(dev_dmu[iGpu]));
	cuda_call(hipFree(dev_dmuxz[iGpu]));

  if(useStreams == 0){
      cuda_call(hipFree(dev_wavefieldVx[iGpu]));
      cuda_call(hipFree(dev_wavefieldVz[iGpu]));
  }
}

void srcAllocateAndCopyToGpu(int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int iGpu){
		// Sources geometry
		cuda_call(hipMalloc((void**) &dev_sourcesPositionRegCenterGrid[iGpu], nSourcesRegCenterGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_sourcesPositionRegCenterGrid[iGpu], sourcesPositionRegCenterGrid, nSourcesRegCenterGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_sourcesPositionRegXGrid[iGpu], nSourcesRegXGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_sourcesPositionRegXGrid[iGpu], sourcesPositionRegXGrid, nSourcesRegXGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_sourcesPositionRegZGrid[iGpu], nSourcesRegZGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_sourcesPositionRegZGrid[iGpu], sourcesPositionRegZGrid, nSourcesRegZGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_sourcesPositionRegXZGrid[iGpu], nSourcesRegXZGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_sourcesPositionRegXZGrid[iGpu], sourcesPositionRegXZGrid, nSourcesRegXZGrid*sizeof(int), hipMemcpyHostToDevice));
}

void recAllocateAndCopyToGpu(int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu){
		// Receivers geometry
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversRegCenterGrid), &nReceiversRegCenterGrid, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_receiversPositionRegCenterGrid[iGpu], nReceiversRegCenterGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_receiversPositionRegCenterGrid[iGpu], receiversPositionRegCenterGrid, nReceiversRegCenterGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversRegXGrid), &nReceiversRegXGrid, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_receiversPositionRegXGrid[iGpu], nReceiversRegXGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_receiversPositionRegXGrid[iGpu], receiversPositionRegXGrid, nReceiversRegXGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversRegZGrid), &nReceiversRegZGrid, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_receiversPositionRegZGrid[iGpu], nReceiversRegZGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_receiversPositionRegZGrid[iGpu], receiversPositionRegZGrid, nReceiversRegZGrid*sizeof(int), hipMemcpyHostToDevice));
		cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversRegXZGrid), &nReceiversRegXZGrid, sizeof(int), 0, hipMemcpyHostToDevice));
		cuda_call(hipMalloc((void**) &dev_receiversPositionRegXZGrid[iGpu], nReceiversRegXZGrid*sizeof(int)));
		cuda_call(hipMemcpy(dev_receiversPositionRegXZGrid[iGpu], receiversPositionRegXZGrid, nReceiversRegXZGrid*sizeof(int), hipMemcpyHostToDevice));
}

void srcRecAllocateAndCopyToGpu(int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu){

		srcAllocateAndCopyToGpu(sourcesPositionRegCenterGrid, nSourcesRegCenterGrid, sourcesPositionRegXGrid, nSourcesRegXGrid, sourcesPositionRegZGrid, nSourcesRegZGrid, sourcesPositionRegXZGrid, nSourcesRegXZGrid, iGpu);
		recAllocateAndCopyToGpu(receiversPositionRegCenterGrid, nReceiversRegCenterGrid, receiversPositionRegXGrid, nReceiversRegXGrid, receiversPositionRegZGrid, nReceiversRegZGrid, receiversPositionRegXZGrid, nReceiversRegXZGrid, iGpu);
}

//allocate source terms on device
void sourceAllocateGpu(int nSourcesRegCenterGrid, int nSourcesRegXGrid, int nSourcesRegZGrid, int nSourcesRegXZGrid, int iGpu){
		cuda_call(hipMalloc((void**) &dev_sourceRegDtw_vx[iGpu], nSourcesRegXGrid*host_ntw*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_sourceRegDtw_vz[iGpu], nSourcesRegZGrid*host_ntw*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_sourceRegDtw_sigmaxx[iGpu], nSourcesRegCenterGrid*host_ntw*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_sourceRegDtw_sigmazz[iGpu], nSourcesRegCenterGrid*host_ntw*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_sourceRegDtw_sigmaxz[iGpu], nSourcesRegXZGrid*host_ntw*sizeof(float)));
}

//copy source terms from host to device
void sourceCopyToGpu(float *sourceRegDtw_vx, float *sourceRegDtw_vz, float *sourceRegDtw_sigmaxx, float *sourceRegDtw_sigmazz, float *sourceRegDtw_sigmaxz, int nSourcesRegCenterGrid, int nSourcesRegXGrid, int nSourcesRegZGrid, int nSourcesRegXZGrid, int iGpu){
		cuda_call(hipMemcpy(dev_sourceRegDtw_vx[iGpu], sourceRegDtw_vx, nSourcesRegXGrid*host_ntw*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_sourceRegDtw_vz[iGpu], sourceRegDtw_vz, nSourcesRegZGrid*host_ntw*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_sourceRegDtw_sigmaxx[iGpu], sourceRegDtw_sigmaxx, nSourcesRegCenterGrid*host_ntw*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_sourceRegDtw_sigmazz[iGpu], sourceRegDtw_sigmazz, nSourcesRegCenterGrid*host_ntw*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_sourceRegDtw_sigmaxz[iGpu], sourceRegDtw_sigmaxz, nSourcesRegXZGrid*host_ntw*sizeof(float), hipMemcpyHostToDevice));
}
//allocate model on device
void dataAllocateGpu(int nReceiversRegCenterGrid, int nReceiversRegXGrid, int nReceiversRegZGrid, int nReceiversRegXZGrid, int iGpu){
		cuda_call(hipMalloc((void**) &dev_dataRegDts_vx[iGpu], nReceiversRegXGrid*host_nts*sizeof(float))); // Allocate output on device
		cuda_call(hipMalloc((void**) &dev_dataRegDts_vz[iGpu], nReceiversRegZGrid*host_nts*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_dataRegDts_sigmaxx[iGpu], nReceiversRegCenterGrid*host_nts*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_dataRegDts_sigmazz[iGpu], nReceiversRegCenterGrid*host_nts*sizeof(float)));
		cuda_call(hipMalloc((void**) &dev_dataRegDts_sigmaxz[iGpu], nReceiversRegXZGrid*host_nts*sizeof(float)));
}
void dataCopyToGpu(float *dataRegDts_vx, float *dataRegDts_vz, float *dataRegDts_sigmaxx, float *dataRegDts_sigmazz, float *dataRegDts_sigmaxz, int nReceiversRegCenterGrid, int nReceiversRegXGrid, int nReceiversRegZGrid, int nReceiversRegXZGrid, int iGpu){
		cuda_call(hipMemcpy(dev_dataRegDts_vx[iGpu], dataRegDts_vx, nReceiversRegXGrid*host_nts*sizeof(float), hipMemcpyHostToDevice)); // Copy rec signals on device
		cuda_call(hipMemcpy(dev_dataRegDts_vz[iGpu], dataRegDts_vz, nReceiversRegZGrid*host_nts*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dataRegDts_sigmaxx[iGpu], dataRegDts_sigmaxx, nReceiversRegCenterGrid*host_nts*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dataRegDts_sigmazz[iGpu], dataRegDts_sigmazz, nReceiversRegCenterGrid*host_nts*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dataRegDts_sigmaxz[iGpu], dataRegDts_sigmaxz, nReceiversRegXZGrid*host_nts*sizeof(float), hipMemcpyHostToDevice));
}
void dataInitializeOnGpu(int nReceiversRegCenterGrid, int nReceiversRegXGrid, int nReceiversRegZGrid, int nReceiversRegXZGrid, int iGpu){
		cuda_call(hipMemset(dev_dataRegDts_vx[iGpu], 0, nReceiversRegXGrid*host_nts*sizeof(float))); // Initialize output on device
		cuda_call(hipMemset(dev_dataRegDts_vz[iGpu], 0, nReceiversRegZGrid*host_nts*sizeof(float))); // Initialize output on device
		cuda_call(hipMemset(dev_dataRegDts_sigmaxx[iGpu], 0, nReceiversRegCenterGrid*host_nts*sizeof(float))); // Initialize output on device
		cuda_call(hipMemset(dev_dataRegDts_sigmazz[iGpu], 0, nReceiversRegCenterGrid*host_nts*sizeof(float))); // Initialize output on device
		cuda_call(hipMemset(dev_dataRegDts_sigmaxz[iGpu], 0, nReceiversRegXZGrid*host_nts*sizeof(float))); // Initialize output on device
}

void wavefieldInitializeOnGpu(int iGpu){
		// Time slices
		cuda_call(hipMemset(dev_p0_vx[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p0_vz[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p0_sigmaxx[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p0_sigmazz[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p0_sigmaxz[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p1_vx[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p1_vz[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p1_sigmaxx[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p1_sigmazz[iGpu], 0, host_nz*host_nx*sizeof(float)));
		cuda_call(hipMemset(dev_p1_sigmaxz[iGpu], 0, host_nz*host_nx*sizeof(float)));
}

void VxVzInitializeOnGpu(int iGpu){
		// Set Vx to zero
		cuda_call(hipMemset(dev_wavefieldVx[iGpu], 0, host_nz*host_nx*host_nts*sizeof(float)));
		// Set Vz to zero
		cuda_call(hipMemset(dev_wavefieldVz[iGpu], 0, host_nz*host_nx*host_nts*sizeof(float)));
}

void modelCopyToGpu(float *drhox, float *drhoz, float *dlame, float *dmu, float *dmuxz,int iGpu){
		cuda_call(hipMemcpy(dev_drhox[iGpu], drhox, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_drhoz[iGpu], drhoz, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dlame[iGpu], dlame, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dmu[iGpu], dmu, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
		cuda_call(hipMemcpy(dev_dmuxz[iGpu], dmuxz, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));
}

void setupBornFwdGpu(float *sourceRegDtw_vx, float *sourceRegDtw_vz, float *sourceRegDtw_sigmaxx, float *sourceRegDtw_sigmazz, float *sourceRegDtw_sigmaxz, float *drhox, float *drhoz, float *dlame, float *dmu, float *dmuxz, float *dataRegDts_vx, float *dataRegDts_vz, float *dataRegDts_sigmaxx, float *dataRegDts_sigmazz, float *dataRegDts_sigmaxz, int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu, int iGpuId){

		// Set device number on GPU cluster
		hipSetDevice(iGpuId);

		//allocate and copy src and rec geometry to gpu
		srcRecAllocateAndCopyToGpu(sourcesPositionRegCenterGrid, nSourcesRegCenterGrid, sourcesPositionRegXGrid, nSourcesRegXGrid, sourcesPositionRegZGrid, nSourcesRegZGrid, sourcesPositionRegXZGrid, nSourcesRegXZGrid, receiversPositionRegCenterGrid, nReceiversRegCenterGrid, receiversPositionRegXGrid, nReceiversRegXGrid, receiversPositionRegZGrid, nReceiversRegZGrid, receiversPositionRegXZGrid, nReceiversRegXZGrid, iGpu);

		// Source - wavelets for each wavefield component. Allocate and copy to gpu
		sourceAllocateGpu(nSourcesRegCenterGrid, nSourcesRegXGrid, nSourcesRegZGrid, nSourcesRegXZGrid, iGpu);
		sourceCopyToGpu(sourceRegDtw_vx, sourceRegDtw_vz, sourceRegDtw_sigmaxx, sourceRegDtw_sigmazz, sourceRegDtw_sigmaxz, nSourcesRegCenterGrid, nSourcesRegXGrid, nSourcesRegZGrid, nSourcesRegXZGrid, iGpu);

		// Data - data recordings for each wavefield component. Allocate and initialize on gpu
		dataAllocateGpu(nReceiversRegCenterGrid, nReceiversRegXGrid, nReceiversRegZGrid, nReceiversRegXZGrid, iGpu);
		dataInitializeOnGpu(nReceiversRegCenterGrid, nReceiversRegXGrid, nReceiversRegZGrid, nReceiversRegXZGrid, iGpu);

		//Initialize wavefield slices to zero
		wavefieldInitializeOnGpu(iGpu);

		//initailize source Vx and Vz to zero
		VxVzInitializeOnGpu(iGpu);

		//Initialize model perturbations
		modelCopyToGpu(drhox,drhoz,dlame,dmu,dmuxz,iGpu);

}

void launchFwdStepKernels(dim3 dimGrid, dim3 dimBlock, int iGpu){
		kernel_exec(ker_step_fwd<<<dimGrid, dimBlock>>>(dev_p0_vx[iGpu], dev_p0_vz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_p0_sigmaxz[iGpu], dev_p1_vx[iGpu], dev_p1_vz[iGpu], dev_p1_sigmaxx[iGpu], dev_p1_sigmazz[iGpu], dev_p1_sigmaxz[iGpu], dev_p0_vx[iGpu], dev_p0_vz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_p0_sigmaxz[iGpu], dev_rhoxDtw[iGpu], dev_rhozDtw[iGpu], dev_lamb2MuDtw[iGpu], dev_lambDtw[iGpu], dev_muxzDtw[iGpu]));
}

void launchFwdInjectSourceKernels(int nSourcesRegCenterGrid, int nSourcesRegXGrid, int nSourcesRegZGrid, int nSourcesRegXZGrid, int itw, int iGpu){

		kernel_exec(ker_inject_source_centerGrid<<<1, nSourcesRegCenterGrid>>>(dev_sourceRegDtw_sigmaxx[iGpu], dev_sourceRegDtw_sigmazz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], itw-1, dev_sourcesPositionRegCenterGrid[iGpu]));

		kernel_exec(ker_inject_source_xGrid<<<1, nSourcesRegXGrid>>>(dev_sourceRegDtw_vx[iGpu], dev_p0_vx[iGpu], itw-1, dev_sourcesPositionRegXGrid[iGpu]));

		kernel_exec(ker_inject_source_zGrid<<<1, nSourcesRegZGrid>>>(dev_sourceRegDtw_vz[iGpu], dev_p0_vz[iGpu], itw-1, dev_sourcesPositionRegZGrid[iGpu]));

		kernel_exec(ker_inject_source_xzGrid<<<1, nSourcesRegXZGrid>>>(dev_sourceRegDtw_sigmaxz[iGpu], dev_p0_sigmaxz[iGpu], itw-1, dev_sourcesPositionRegXZGrid[iGpu]));

}

void launchDampCosineEdgeKernels(dim3 dimGrid, dim3 dimBlock, int iGpu){
		kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0_vx[iGpu], dev_p1_vx[iGpu], dev_p0_vz[iGpu],  dev_p1_vz[iGpu], dev_p0_sigmaxx[iGpu], dev_p1_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_p1_sigmazz[iGpu], dev_p0_sigmaxz[iGpu], dev_p1_sigmaxz[iGpu]));
}

void switchPointers(int iGpu){
		dev_temp1[iGpu] = dev_p0_vx[iGpu];
		dev_p0_vx[iGpu] = dev_p1_vx[iGpu];
		dev_p1_vx[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] = dev_p0_vz[iGpu];
		dev_p0_vz[iGpu] = dev_p1_vz[iGpu];
		dev_p1_vz[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] = dev_p0_sigmaxx[iGpu];
		dev_p0_sigmaxx[iGpu] = dev_p1_sigmaxx[iGpu];
		dev_p1_sigmaxx[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] = dev_p0_sigmazz[iGpu];
		dev_p0_sigmazz[iGpu] = dev_p1_sigmazz[iGpu];
		dev_p1_sigmazz[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] = dev_p0_sigmaxz[iGpu];
		dev_p0_sigmaxz[iGpu] = dev_p1_sigmaxz[iGpu];
		dev_p1_sigmaxz[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] = NULL;
}

void switchPointersSecondarySource(int iGpu){
		dev_temp1[iGpu] 	= dev_ssVxLeft[iGpu];
		dev_ssVxLeft[iGpu] 	= dev_ssVxRight[iGpu];
		dev_ssVxRight[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] 	= dev_ssVzLeft[iGpu];
		dev_ssVzLeft[iGpu] 	= dev_ssVzRight[iGpu];
		dev_ssVzRight[iGpu] = dev_temp1[iGpu];

		dev_temp1[iGpu] 		= dev_ssSigmaxxLeft[iGpu];
		dev_ssSigmaxxLeft[iGpu] = dev_ssSigmaxxRight[iGpu];
		dev_ssSigmaxxRight[iGpu]= dev_temp1[iGpu];

		dev_temp1[iGpu] 		= dev_ssSigmazzLeft[iGpu];
		dev_ssSigmazzLeft[iGpu] = dev_ssSigmazzRight[iGpu];
		dev_ssSigmazzRight[iGpu]= dev_temp1[iGpu];

		dev_temp1[iGpu] 		= dev_ssSigmaxzLeft[iGpu];
		dev_ssSigmaxzLeft[iGpu] = dev_ssSigmaxzRight[iGpu];
		dev_ssSigmaxzRight[iGpu]= dev_temp1[iGpu];

		dev_temp1[iGpu] = NULL;
}

void SecondarySourceInitializeOnGpu(int iGpu){
	//Born specific slices
	cuda_call(hipMemset(dev_ssVxLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssVxRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssVzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssVzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmaxxLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmaxxRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmazzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmazzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmaxzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_ssSigmaxzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
}

void launchFwdRecordInterpDataKernels(int nblockDataCenterGrid, int nblockDataXGrid, int nblockDataZGrid, int nblockDataXZGrid, int its, int it2, int iGpu){
		kernel_exec(ker_record_interp_data_centerGrid<<<nblockDataCenterGrid, BLOCK_SIZE_DATA>>>( dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_dataRegDts_sigmaxx[iGpu], dev_dataRegDts_sigmazz[iGpu], its, it2, dev_receiversPositionRegCenterGrid[iGpu]));
		kernel_exec(ker_record_interp_data_xGrid<<<nblockDataXGrid, BLOCK_SIZE_DATA>>>( dev_p0_vx[iGpu], dev_dataRegDts_vx[iGpu], its, it2, dev_receiversPositionRegXGrid[iGpu]));

		kernel_exec(ker_record_interp_data_zGrid<<<nblockDataZGrid, BLOCK_SIZE_DATA>>>( dev_p0_vz[iGpu], dev_dataRegDts_vz[iGpu], its, it2, dev_receiversPositionRegZGrid[iGpu]));

		kernel_exec(ker_record_interp_data_xzGrid<<<nblockDataXZGrid, BLOCK_SIZE_DATA>>>( dev_p0_sigmaxz[iGpu], dev_dataRegDts_sigmaxz[iGpu], its, it2, dev_receiversPositionRegXZGrid[iGpu]));
}

void modelSetOnGpu(int iGpu){
  cuda_call(hipMemset(dev_drhox[iGpu], 0, host_nz*host_nx*sizeof(float)));
  cuda_call(hipMemset(dev_drhoz[iGpu], 0, host_nz*host_nx*sizeof(float)));
  cuda_call(hipMemset(dev_dlame[iGpu], 0, host_nz*host_nx*sizeof(float)));
  cuda_call(hipMemset(dev_dmu[iGpu], 0, host_nz*host_nx*sizeof(float)));
  cuda_call(hipMemset(dev_dmuxz[iGpu], 0, host_nz*host_nx*sizeof(float)));
}

//Functions specific to Adjoing operator
void setupAdjGpu(float *sourceRegDtw_vx, float *sourceRegDtw_vz, float *sourceRegDtw_sigmaxx, float *sourceRegDtw_sigmazz, float *sourceRegDtw_sigmaxz, float *dataRegDts_vx, float *dataRegDts_vz, float *dataRegDts_sigmaxx, float *dataRegDts_sigmazz, float *dataRegDts_sigmaxz, int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu, int iGpuId){
		// Set device number on GPU cluster
		hipSetDevice(iGpuId);

	//allocate and copy src and rec geometry to gpu
		srcRecAllocateAndCopyToGpu(sourcesPositionRegCenterGrid, nSourcesRegCenterGrid, sourcesPositionRegXGrid, nSourcesRegXGrid, sourcesPositionRegZGrid, nSourcesRegZGrid, sourcesPositionRegXZGrid, nSourcesRegXZGrid, receiversPositionRegCenterGrid, nReceiversRegCenterGrid, receiversPositionRegXGrid, nReceiversRegXGrid, receiversPositionRegZGrid, nReceiversRegZGrid, receiversPositionRegXZGrid, nReceiversRegXZGrid, iGpu);

		// Source - wavelets for each wavefield component. Allocate and copy to gpu
		sourceAllocateGpu(nSourcesRegCenterGrid, nSourcesRegXGrid, nSourcesRegZGrid, nSourcesRegXZGrid, iGpu);
		sourceCopyToGpu(sourceRegDtw_vx, sourceRegDtw_vz, sourceRegDtw_sigmaxx, sourceRegDtw_sigmazz, sourceRegDtw_sigmaxz, nSourcesRegCenterGrid, nSourcesRegXGrid, nSourcesRegZGrid, nSourcesRegXZGrid, iGpu);

		// Data - data recordings for each wavefield component. Allocate and initialize on gpu
		dataAllocateGpu(nReceiversRegCenterGrid, nReceiversRegXGrid, nReceiversRegZGrid, nReceiversRegXZGrid, iGpu);
		dataCopyToGpu(dataRegDts_vx, dataRegDts_vz, dataRegDts_sigmaxx, dataRegDts_sigmazz, dataRegDts_sigmaxz, nReceiversRegCenterGrid, nReceiversRegXGrid, nReceiversRegZGrid, nReceiversRegXZGrid, iGpu);

		//initailize wavefield slices to zero
		wavefieldInitializeOnGpu(iGpu);

		//initailize source Vx and Vz to zero
		VxVzInitializeOnGpu(iGpu);

    //Initialize model perturbations to zero
    modelSetOnGpu(iGpu);
}

void launchAdjStepKernels(dim3 dimGrid, dim3 dimBlock, int iGpu){
		kernel_exec(ker_step_adj<<<dimGrid, dimBlock>>>(dev_p0_vx[iGpu], dev_p0_vz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_p0_sigmaxz[iGpu], dev_p1_vx[iGpu], dev_p1_vz[iGpu], dev_p1_sigmaxx[iGpu], dev_p1_sigmazz[iGpu], dev_p1_sigmaxz[iGpu], dev_p0_vx[iGpu], dev_p0_vz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], dev_p0_sigmaxz[iGpu], dev_rhoxDtw[iGpu], dev_rhozDtw[iGpu], dev_lamb2MuDtw[iGpu], dev_lambDtw[iGpu], dev_muxzDtw[iGpu]));
}

void launchAdjInterpInjectDataKernels(int nblockDataCenterGrid, int nblockDataXGrid, int nblockDataZGrid, int nblockDataXZGrid, int its, int it2, int iGpu){
		kernel_exec(ker_interp_inject_data_centerGrid<<<nblockDataCenterGrid, BLOCK_SIZE_DATA>>>( dev_dataRegDts_sigmaxx[iGpu], dev_dataRegDts_sigmazz[iGpu], dev_p0_sigmaxx[iGpu], dev_p0_sigmazz[iGpu], its, it2, dev_receiversPositionRegCenterGrid[iGpu]));

		kernel_exec(ker_interp_inject_data_xGrid<<<nblockDataXGrid, BLOCK_SIZE_DATA>>>(dev_dataRegDts_vx[iGpu], dev_p0_vx[iGpu], its, it2, dev_receiversPositionRegXGrid[iGpu]));


		kernel_exec(ker_interp_inject_data_zGrid<<<nblockDataZGrid, BLOCK_SIZE_DATA>>>(dev_dataRegDts_vz[iGpu], dev_p0_vz[iGpu], its, it2, dev_receiversPositionRegZGrid[iGpu]));
		kernel_exec(ker_interp_inject_data_xzGrid<<<nblockDataXZGrid, BLOCK_SIZE_DATA>>>(dev_dataRegDts_sigmaxz[iGpu], dev_p0_sigmaxz[iGpu], its, it2, dev_receiversPositionRegXZGrid[iGpu]));

}

/****************************************************************************************/
/*********************************** Born forward operator ******************************/
/****************************************************************************************/
void BornShotsFwdGpu(float *sourceRegDtw_vx, float *sourceRegDtw_vz, float *sourceRegDtw_sigmaxx, float *sourceRegDtw_sigmazz, float *sourceRegDtw_sigmaxz, float *drhox, float *drhoz, float *dlame, float *dmu, float *dmuxz, float *dataRegDts_vx, float *dataRegDts_vz, float *dataRegDts_sigmaxx, float *dataRegDts_sigmazz, float *dataRegDts_sigmaxz, int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu, int iGpuId, int surfaceCondition, int useStreams){
    //setup:                a) src and receiver positions allocation and copying to device
    //                      b) allocate and copy model (arrays for sources for each wavefield) to device
    //                      c) allocate and initialize(0) data (recevier recordings arrays) to device
    //                      d) allocate and copy wavefield time slices to gpu

    setupBornFwdGpu(sourceRegDtw_vx, sourceRegDtw_vz, sourceRegDtw_sigmaxx, sourceRegDtw_sigmazz, sourceRegDtw_sigmaxz, drhox, drhoz, dlame, dmu, dmuxz, dataRegDts_vx, dataRegDts_vz, dataRegDts_sigmaxx, dataRegDts_sigmazz, dataRegDts_sigmaxz, sourcesPositionRegCenterGrid, nSourcesRegCenterGrid, sourcesPositionRegXGrid, nSourcesRegXGrid, sourcesPositionRegZGrid, nSourcesRegZGrid, sourcesPositionRegXZGrid, nSourcesRegXZGrid, receiversPositionRegCenterGrid, nReceiversRegCenterGrid, receiversPositionRegXGrid, nReceiversRegXGrid, receiversPositionRegZGrid, nReceiversRegZGrid, receiversPositionRegXZGrid, nReceiversRegXZGrid, iGpu, iGpuId);

	//Finite-difference grid and blocks
	int nblockx;
	if(surfaceCondition==1){
		// Free-surface boundary condition
		nblockx = (host_nz-5-FAT) / BLOCK_SIZE;
	}
	else if(surfaceCondition==0){
		nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	}
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Extraction grid size
	int nblockDataCenterGrid = (nReceiversRegCenterGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataXGrid = (nReceiversRegXGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataZGrid = (nReceiversRegZGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataXZGrid = (nReceiversRegXZGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	if(useStreams == 0){

		//Born operator w/o the use of Streams
		/************************** Source wavefield computation ****************************/
		for (int its = 0; its < host_nts-1; its++){
				for (int it2 = 1; it2 < host_sub+1; it2++){
						// Compute fine time-step index
						int itw = its * host_sub + it2;

						// Step forward
						launchFwdStepKernels(dimGrid, dimBlock, iGpu);

						// Inject source
						launchFwdInjectSourceKernels(nSourcesRegCenterGrid,nSourcesRegXGrid,nSourcesRegZGrid,nSourcesRegXZGrid, itw, iGpu);

						// Damp wavefields
						launchDampCosineEdgeKernels(dimGrid, dimBlock, iGpu);

						// Extract wavefield components
						kernel_exec(interpWavefieldSingleComp<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_p0_vx[iGpu], its, it2));
						kernel_exec(interpWavefieldSingleComp<<<dimGrid, dimBlock>>>(dev_wavefieldVz[iGpu], dev_p0_vz[iGpu], its, it2));

						// Switch pointers
						switchPointers(iGpu);

				}
		}

		/************************** Scattered wavefield computation *************************/
		// Initialize time slices on device
		wavefieldInitializeOnGpu(iGpu);
	  SecondarySourceInitializeOnGpu(iGpu);

		// Copy model perturbations to device (done within setupBornFwdGpu function)
		//Note the perturbations have been already scaled by the wave-equation source scaling factor outside of this function

		// Compute secondary source for first coarse time index (its = 0)
		kernel_exec(imagingElaFwdGpu<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_wavefieldVz[iGpu], dev_ssVxLeft[iGpu], dev_ssVzLeft[iGpu], dev_ssSigmaxxLeft[iGpu], dev_ssSigmazzLeft[iGpu], dev_ssSigmaxzLeft[iGpu], dev_drhox[iGpu], dev_drhoz[iGpu], dev_dlame[iGpu], dev_dmu[iGpu], dev_dmuxz[iGpu], 0));

		// Start propagating scattered wavefield
		for (int its = 0; its < host_nts-1; its++){

			// Compute secondary source for first coarse time index (its+1)
			kernel_exec(imagingElaFwdGpu<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_wavefieldVz[iGpu], dev_ssVxRight[iGpu], dev_ssVzRight[iGpu], dev_ssSigmaxxRight[iGpu], dev_ssSigmazzRight[iGpu], dev_ssSigmaxzRight[iGpu], dev_drhox[iGpu], dev_drhoz[iGpu], dev_dlame[iGpu], dev_dmu[iGpu], dev_dmuxz[iGpu], its+1));

			for (int it2 = 1; it2 < host_sub+1; it2++){

				// Step forward
				launchFwdStepKernels(dimGrid, dimBlock, iGpu);

				// Inject secondary source sample itw-1 in each component
				kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssVxLeft[iGpu], dev_ssVxRight[iGpu], dev_p0_vx[iGpu], it2-1));
				kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssVzLeft[iGpu], dev_ssVzRight[iGpu], dev_p0_vz[iGpu], it2-1));
				kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssSigmaxxLeft[iGpu], dev_ssSigmaxxRight[iGpu], dev_p0_sigmaxx[iGpu], it2-1));
				kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssSigmazzLeft[iGpu], dev_ssSigmazzRight[iGpu], dev_p0_sigmazz[iGpu], it2-1));
				kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssSigmaxzLeft[iGpu], dev_ssSigmaxzRight[iGpu], dev_p0_sigmaxz[iGpu], it2-1));

				// Damp wavefields
				launchDampCosineEdgeKernels(dimGrid, dimBlock, iGpu);

				// Extract data
				launchFwdRecordInterpDataKernels(nblockDataCenterGrid, nblockDataXGrid, nblockDataZGrid, nblockDataXZGrid, its, it2, iGpu);

				// Switch pointers
				switchPointers(iGpu);

			}

			// Switch pointers for secondary source and setting right slices to zero
			switchPointersSecondarySource(iGpu);
			cuda_call(hipMemset(dev_ssVxRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
			cuda_call(hipMemset(dev_ssVzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
			cuda_call(hipMemset(dev_ssSigmaxxRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
			cuda_call(hipMemset(dev_ssSigmazzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));
			cuda_call(hipMemset(dev_ssSigmaxzRight[iGpu], 0, host_nz*host_nx*sizeof(float)));

		}

		// Copy data back to host
		cuda_call(hipMemcpy(dataRegDts_vx, dev_dataRegDts_vx[iGpu], nReceiversRegXGrid*host_nts*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dataRegDts_vz, dev_dataRegDts_vz[iGpu], nReceiversRegZGrid*host_nts*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dataRegDts_sigmaxx, dev_dataRegDts_sigmaxx[iGpu], nReceiversRegCenterGrid*host_nts*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dataRegDts_sigmazz, dev_dataRegDts_sigmazz[iGpu], nReceiversRegCenterGrid*host_nts*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dataRegDts_sigmaxz, dev_dataRegDts_sigmaxz[iGpu], nReceiversRegXZGrid*host_nts*sizeof(float), hipMemcpyDeviceToHost));

		// Deallocate all slices
		cuda_call(hipFree(dev_sourceRegDtw_vx[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_vz[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmaxx[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmazz[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmaxz[iGpu]));

		cuda_call(hipFree(dev_dataRegDts_vx[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_vz[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmaxx[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmazz[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmaxz[iGpu]));

		cuda_call(hipFree(dev_sourcesPositionRegCenterGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegXGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegZGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegXZGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegCenterGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegXGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegZGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegXZGrid[iGpu]));

	} else {
		//Born operator w/ the use of Streams
		throw std::logic_error( "Error! Born forward operator not implemented yet!" );

	}
}

/****************************************************************************************/
/*********************************** Born adjoint operator ******************************/
/****************************************************************************************/
void BornShotsAdjGpu(float *sourceRegDtw_vx, float *sourceRegDtw_vz, float *sourceRegDtw_sigmaxx, float *sourceRegDtw_sigmazz, float *sourceRegDtw_sigmaxz, float *drhox, float *drhoz, float *dlame, float *dmu, float *dmuxz, float *dataRegDts_vx, float *dataRegDts_vz, float *dataRegDts_sigmaxx, float *dataRegDts_sigmazz, float *dataRegDts_sigmaxz, int *sourcesPositionRegCenterGrid, int nSourcesRegCenterGrid, int *sourcesPositionRegXGrid, int nSourcesRegXGrid, int *sourcesPositionRegZGrid, int nSourcesRegZGrid, int *sourcesPositionRegXZGrid, int nSourcesRegXZGrid, int *receiversPositionRegCenterGrid, int nReceiversRegCenterGrid, int *receiversPositionRegXGrid, int nReceiversRegXGrid, int *receiversPositionRegZGrid, int nReceiversRegZGrid, int *receiversPositionRegXZGrid, int nReceiversRegXZGrid, int iGpu, int iGpuId, int surfaceCondition, int useStreams){

	//setup:                a) src and receiver positions allocation and copying to device
	//                      b) allocate and initialize (0) model (arrays for sources for each wavefield) to device
	//                      c) allocate and copy data (recevier recordings arrays) to device
	//                      d) allocate and copy wavefield time slices to gpu
	setupAdjGpu(sourceRegDtw_vx, sourceRegDtw_vz, sourceRegDtw_sigmaxx, sourceRegDtw_sigmazz, sourceRegDtw_sigmaxz, dataRegDts_vx, dataRegDts_vz, dataRegDts_sigmaxx, dataRegDts_sigmazz, dataRegDts_sigmaxz, sourcesPositionRegCenterGrid, nSourcesRegCenterGrid, sourcesPositionRegXGrid, nSourcesRegXGrid, sourcesPositionRegZGrid, nSourcesRegZGrid, sourcesPositionRegXZGrid, nSourcesRegXZGrid, receiversPositionRegCenterGrid, nReceiversRegCenterGrid, receiversPositionRegXGrid, nReceiversRegXGrid, receiversPositionRegZGrid, nReceiversRegZGrid, receiversPositionRegXZGrid, nReceiversRegXZGrid, iGpu, iGpuId);

	//Initialize model perturbations (for allowing add = true from calling function)
	modelCopyToGpu(drhox,drhoz,dlame,dmu,dmuxz,iGpu);

	//Finite-difference grid and blocks
	int nblockx;
	if(surfaceCondition==1){
		// Free-surface boundary condition
		nblockx = (host_nz-5-FAT) / BLOCK_SIZE;
	}
	else if(surfaceCondition==0){
		nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	}
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Extraction grid size
	int nblockDataCenterGrid = (nReceiversRegCenterGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataXGrid = (nReceiversRegXGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataZGrid = (nReceiversRegZGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	int nblockDataXZGrid = (nReceiversRegXZGrid+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	if(useStreams == 0){
		//Born operator w/o the use of Streams
		/************************** Source wavefield computation ****************************/
		for (int its = 0; its < host_nts-1; its++){
				for (int it2 = 1; it2 < host_sub+1; it2++){
						// Compute fine time-step index
						int itw = its * host_sub + it2;

						// Step forward
						launchFwdStepKernels(dimGrid, dimBlock, iGpu);

						// Inject source
						launchFwdInjectSourceKernels(nSourcesRegCenterGrid,nSourcesRegXGrid,nSourcesRegZGrid,nSourcesRegXZGrid, itw, iGpu);

						// Damp wavefields
						launchDampCosineEdgeKernels(dimGrid, dimBlock, iGpu);

						// Extract wavefield components
						kernel_exec(interpWavefieldSingleComp<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_p0_vx[iGpu], its, it2));
						kernel_exec(interpWavefieldSingleComp<<<dimGrid, dimBlock>>>(dev_wavefieldVz[iGpu], dev_p0_vz[iGpu], its, it2));

						// Switch pointers
						switchPointers(iGpu);
				}
		}

		/************************** Receiver wavefield computation **************************/
		// Initialize time slices on device
		wavefieldInitializeOnGpu(iGpu);
		SecondarySourceInitializeOnGpu(iGpu);

	    // Start propagation
	    for (int its = host_nts-2; its > -1; its--){
	        for (int it2 = host_sub-1; it2 > -1; it2--){
	            // Step back in time
	            launchAdjStepKernels(dimGrid, dimBlock, iGpu);
	            // Inject data
	            launchAdjInterpInjectDataKernels(nblockDataCenterGrid, nblockDataXGrid, nblockDataZGrid, nblockDataXZGrid, its, it2, iGpu);
	            // Damp wavefield
	            launchDampCosineEdgeKernels(dimGrid, dimBlock, iGpu);
	            // Interpolate and record time slices of receiver wavefield at coarse sampling (no scaling applied yet)
							kernel_exec(extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssVxLeft[iGpu], dev_ssVxRight[iGpu], dev_p0_vx[iGpu], it2));
							kernel_exec(extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssVzLeft[iGpu], dev_ssVzRight[iGpu], dev_p0_vz[iGpu], it2));
							kernel_exec(extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssSigmaxxLeft[iGpu], dev_ssSigmaxxRight[iGpu], dev_p0_sigmaxx[iGpu], it2));
							kernel_exec(extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssSigmazzLeft[iGpu], dev_ssSigmazzRight[iGpu], dev_p0_sigmazz[iGpu], it2));
							kernel_exec(extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssSigmaxzLeft[iGpu], dev_ssSigmaxzRight[iGpu], dev_p0_sigmaxz[iGpu], it2));

	            // Switch pointers
	            switchPointers(iGpu);
	          }

			  // Apply extended imaging condition for its+1
			  kernel_exec(imagingElaAdjGpu<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_wavefieldVz[iGpu], dev_ssVxRight[iGpu], dev_ssVzRight[iGpu], dev_ssSigmaxxRight[iGpu], dev_ssSigmazzRight[iGpu], dev_ssSigmaxzRight[iGpu], dev_drhox[iGpu], dev_drhoz[iGpu], dev_dlame[iGpu], dev_dmu[iGpu], dev_dmuxz[iGpu], its+1));
			  // Switch pointers for secondary source and setting right slices to zero
			  switchPointersSecondarySource(iGpu);
			  cuda_call(hipMemset(dev_ssVxLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
			  cuda_call(hipMemset(dev_ssVzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
			  cuda_call(hipMemset(dev_ssSigmaxxLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
			  cuda_call(hipMemset(dev_ssSigmazzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
			  cuda_call(hipMemset(dev_ssSigmaxzLeft[iGpu], 0, host_nz*host_nx*sizeof(float)));
	    }// Finished main loop - we still have to compute imaging condition for its=0

		// Apply extended imaging condition for its=0
		kernel_exec(imagingElaAdjGpu<<<dimGrid, dimBlock>>>(dev_wavefieldVx[iGpu], dev_wavefieldVz[iGpu], dev_ssVxRight[iGpu], dev_ssVzRight[iGpu], dev_ssSigmaxxRight[iGpu], dev_ssSigmazzRight[iGpu], dev_ssSigmaxzRight[iGpu], dev_drhox[iGpu], dev_drhoz[iGpu], dev_dlame[iGpu], dev_dmu[iGpu], dev_dmuxz[iGpu], 0));

		// Copy model back to host
		cuda_call(hipMemcpy(drhox, dev_drhox[iGpu], host_nz*host_nx*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(drhoz, dev_drhoz[iGpu], host_nz*host_nx*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dlame, dev_dlame[iGpu], host_nz*host_nx*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dmu, dev_dmu[iGpu], host_nz*host_nx*sizeof(float), hipMemcpyDeviceToHost));
		cuda_call(hipMemcpy(dmuxz, dev_dmuxz[iGpu], host_nz*host_nx*sizeof(float), hipMemcpyDeviceToHost));

		// Deallocate all slices
		cuda_call(hipFree(dev_sourceRegDtw_vx[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_vz[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmaxx[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmazz[iGpu]));
		cuda_call(hipFree(dev_sourceRegDtw_sigmaxz[iGpu]));

		cuda_call(hipFree(dev_dataRegDts_vx[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_vz[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmaxx[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmazz[iGpu]));
		cuda_call(hipFree(dev_dataRegDts_sigmaxz[iGpu]));

		cuda_call(hipFree(dev_sourcesPositionRegCenterGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegXGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegZGrid[iGpu]));
		cuda_call(hipFree(dev_sourcesPositionRegXZGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegCenterGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegXGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegZGrid[iGpu]));
		cuda_call(hipFree(dev_receiversPositionRegXZGrid[iGpu]));

	} else {
		//Born operator w/ the use of Streams
		throw std::logic_error( "Error! Born adjoint operator not implemented yet!" );

	}
}
